#include "hip/hip_runtime.h"
#include "CUDAHandler.h"


constexpr int MAX_RADIUS = 12;                    // change to suit your max
constexpr int MAX_K      = (2*MAX_RADIUS+1)*(2*MAX_RADIUS+1);

__constant__ float d_kernelConst[MAX_K];          // <── NOT a pointer



__device__ float random_float_in_range(hiprandState_t* state, float a, float b) {
    // return a + (b - a) * curand_uniform_float(state);  // this does not include b  e.g -1 to 1.0  it does not include 1.0
    return a + (b - a) * (hiprand_uniform(state) - 0.5) * 2.0;  // this approach includes the upper limit   -1 to 1.0  it includes 1.0
}

__device__
void drawPixel(hipSurfaceObject_t surface, int x, int y, uchar4 color, int width, int height)
{
    if (x >= 0 && x < width && y >= 0 && y < height) {
        surf2Dwrite(color, surface, x * sizeof(uchar4), y);
    }
}

__device__ void drawFilledCircle(hipSurfaceObject_t surface, int cx, int cy, int radius, uchar4 color, int width, int height) {
    int rSquared = radius * radius;
    for (int dy = -radius; dy <= radius; ++dy) {
        int y = cy + dy;
        if (y < 0 || y >= height) continue;

        for (int dx = -radius; dx <= radius; ++dx) {
            int x = cx + dx;
            if (x < 0 || x >= width) continue;

            if (dx * dx + dy * dy <= rSquared) {
                drawPixel(surface, x, y, color, width, height);
            }
        }
    }
}


__global__ void init_random(unsigned int seed, hiprandState_t* states){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    hiprand_init(seed, idx, 0, &states[idx]);
}
__global__ void initializeLeniaParticle(Particle* particles, int totalParticles, hiprandState_t* states, uchar4* colors, int numberColors) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i >= totalParticles) return;
    hiprandState_t x = states[i];
    // float e =  random_float_in_range(&x, 0.1, 0.5);
    // initial noise
    float e = hiprand_uniform(&x) < 0.15f ? hiprand_uniform(&x) * 0.35f : 0.0f; 
    particles[i].energy = e;
    states[i] = x; // reinstate the random value;
    int colorIndex = static_cast<int>(e * (numberColors - 1));
    colorIndex = max(0, min(colorIndex, numberColors - 1));
    particles[i].color = colors[colorIndex];
}

// __global__ void drawLeniaParticles(hipSurfaceObject_t surface, Particle* particles, int numberParticles, int width, int height, float zoom, float panX, float panY){
//     int i = blockIdx.x * blockDim.x + threadIdx.x;
//     if (i >= numberParticles) return;
//     Particle p = particles[i];
//     vec2 pos = p.position;
//     // if (threadIdx.x == 0){
//     //     printf("position.x  %f\n", pos.x);
//     //     printf("Energy %f\n", p.energy);
//     // }
//     float radius = p.radius * zoom;
//     int x0 = (int)(width / 2.0f + (pos.x + panX) * zoom);
//     int y0 = (int)(height / 2.0f + (pos.y + panY) * zoom);
    
//     drawFilledCircle(surface, x0, y0, radius, p.color, width, height);
// }

__global__ void commitNextEnergy_kernel(Particle* particles, int totalParticles) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i >= totalParticles) return;

    particles[i].energy = particles[i].nextEnergy;
    particles[i].nextEnergy = 0;
}

__global__ void thresholdAndCommit_kernel(Particle* g,
                                          int n,
                                          const uchar4* colors,
                                          int numColors,
                                          float thresh=0.3f /* e.g. 0.3f */)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i >= n) return;

    const float e = g[i].energy;          // already up-to-date from step-2

    // 1.  Boolean life flag (only used if you still need a discrete state)
    g[i].alive = (e > thresh);

    // 2.  Colour lookup for rendering – linear ramp through your palette
    int idx = (int)(e * (numColors - 1) + 0.5f);
    idx = max(0, min(idx, numColors - 1));
    g[i].color = colors[idx];
}

__device__ float growthMapping(float u, float mu, float sigma) {
    return 2.0f * expf(-powf((u - mu), 2) / (2.0f * sigma * sigma)) - 1.0f;
}

__global__ void activate_LeniaGoL_convolution_kernel(
    Particle* particles,
    int totalParticles,
    int gridRows,
    int gridCols,
    int kernelDiameter,
    int radius,
    float sigma,
    float mu,
    float dt
) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i >= totalParticles) return;

    int row = i / gridCols;
    int col = i % gridCols;

    float neighborSum = 0.0f;
    float weightSum = 0.0f;

    for (int dr = -radius; dr <= radius; ++dr) {
        for (int dc = -radius; dc <= radius; ++dc) {
            int nr = row + dr;
            int nc = col + dc;
            // float distance = sqrtf(dr * dr + dc * dc);
            if (nr >= 0 && nr < gridRows && nc >= 0 && nc < gridCols) {
                int j = nr * gridCols + nc;
                // float weight = gaussianKernel(distance / radius, sigma);
                int kernelIndex = (dr + radius) * kernelDiameter + (dc + radius);
                float weight = d_kernelConst[kernelIndex];
                neighborSum += particles[j].energy * weight;
                weightSum += weight;
            }
        }
    }

   
    float u = neighborSum / weightSum;
    

    float growth = growthMapping(u, mu, sigma);
    float e = fminf(1.0f, fmaxf(0.0f, particles[i].energy + dt * growth));
    particles[i].nextEnergy = e;
}



__global__ void drawLeniaParticles(hipSurfaceObject_t surface, Particle* particles, int numberParticles, int width, int height, float zoom, float panX, float panY){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= numberParticles) return;

    Particle p = particles[i];

    // Apply zoom and pan directly (without adding screen center shift)
    float radius = p.radius * zoom;
    int x0 = (int)((p.position.x + panX) * zoom);
    int y0 = (int)((p.position.y + panY) * zoom);

    // Reject particles that fall outside the surface
    if (x0 < 0 || x0 >= width || y0 < 0 || y0 >= height) return;

    drawFilledCircle(surface, x0, y0, radius, p.color, width, height);
}


__global__ void clearSurface_kernel(hipSurfaceObject_t surface, int width, int height, uchar4 color) {
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;

    if (x >= width || y >= height) return;
    surf2Dwrite(color, surface, x * sizeof(uchar4), y);
}



// __device__
// void drawPixel(hipSurfaceObject_t surface, int x, int y, uchar4 color, int width, int height)
// {
//     if (x >= 0 && x < width && y >= 0 && y < height) {
//         surf2Dwrite(color, surface, x * sizeof(uchar4), y);
//     }
// }

__device__ void drawLine(hipSurfaceObject_t surface, int x0, int y0, int x1, int y1, uchar4 color, int width, int height)
{
    int dx = abs(x1 - x0), sx = x0 < x1 ? 1 : -1;
    int dy = -abs(y1 - y0), sy = y0 < y1 ? 1 : -1;
    int err = dx + dy, e2;

    while (true) {
        drawPixel(surface, x0, y0, color, width, height);
        if (x0 == x1 && y0 == y1) break;
        e2 = 2 * err;
        if (e2 >= dy) { err += dy; x0 += sx; }
        if (e2 <= dx) { err += dx; y0 += sy; }
    }
}

__device__ void drawCircleOutline(hipSurfaceObject_t surface, int cx, int cy, int radius, uchar4 color, int width, int height) {
    const int segments = 36; // More segments = smoother circle
    for (int i = 0; i < segments; ++i) {
        float theta0 = (2.0f * M_PI * i) / segments;
        float theta1 = (2.0f * M_PI * (i + 1)) / segments;
        
        int x0 = cx + radius * cosf(theta0);
        int y0 = cy + radius * sinf(theta0);
        int x1 = cx + radius * cosf(theta1);
        int y1 = cy + radius * sinf(theta1);

        drawLine(surface, x0, y0, x1, y1, color, width, height);
    }
}

// __device__ void drawFilledCircle(hipSurfaceObject_t surface, int cx, int cy, int radius, uchar4 color, int width, int height) {
//     int rSquared = radius * radius;
//     for (int dy = -radius; dy <= radius; ++dy) {
//         int y = cy + dy;
//         if (y < 0 || y >= height) continue;

//         for (int dx = -radius; dx <= radius; ++dx) {
//             int x = cx + dx;
//             if (x < 0 || x >= width) continue;

//             if (dx * dx + dy * dy <= rSquared) {
//                 drawPixel(surface, x, y, color, width, height);
//             }
//         }
//     }
// }

__device__ bool threeEdgeTest2D(const vec2 &A, const vec2 &B, const vec2 &C, const vec2 &P)
{
    vec2 a = B - A;
    vec2 b = C - A;
    vec2 q = P - A;

    float cross_ab = a.x * b.y - a.y * b.x;

    if (cross_ab == 0.0f) return false; // Triangle is degenerate (no area)

    float s = (q.x * b.y - q.y * b.x) / cross_ab;
    float r = (a.x * q.y - a.y * q.x) / cross_ab;

    return s >= 0.0f && r >= 0.0f && (s + r) <= 1.0f;
}

__device__ __host__ void swap(vec2& a, vec2& b) {
        vec2 temp = a;
        a = b;
        b = temp;
}


__global__ void drawTriangle_kernel(hipSurfaceObject_t surface, int width, int height, uchar4 color, int minX, int minY, vec2 a, vec2 b, vec2 c)
{
    int local_x = threadIdx.x + blockIdx.x * blockDim.x;
    int local_y = threadIdx.y + blockIdx.y * blockDim.y;

    int x = local_x + minX;
    int y = local_y + minY;

    if (x >= width || y >= height) return;

    if (!threeEdgeTest2D(a, b, c, vec2(x, y))) return;
    drawPixel(surface, x, y, color, width, height);

    
}


// ────────────────────────────────
//  Chan-Lenia “shell” kernel
//  (build once on the host)
// ────────────────────────────────
std::vector<float> CUDAHandler::generateCircularShellKernel(int radius, float alpha /* ≈4 */)
{
    int  diameter = 2 * radius + 1;
    std::vector<float> kernel(diameter * diameter, 0.0f);

    float sum = 0.0f;
    for (int y = -radius; y <= radius; ++y)
        for (int x = -radius; x <= radius; ++x)
        {
            float dist = std::sqrt(float(x*x + y*y));     // Euclidean distance
            if (dist > radius) continue;                  // outside the circle

            float r = dist / radius;                      // normalise to [0,1]
            float value = std::exp(alpha - alpha / (4.0f * r * (1.0f - r)));
            // ────────────────▲  shell(r, alpha)
            // float value = std::exp(-(r * r) / (2.0f * alpha * alpha));
            // float value = std::exp(alpha - (alpha/(4 * r * (1 - r))));
            // float value = std::pow(4 * r * (1 - r), alpha);
            // float value = std::exp(alpha -1.0f / (2.0f * r * r));

            kernel[(y + radius) * diameter + (x + radius)] = value;
            sum += value;
        }

    // normalise so ΣK = 1 (preserves total mass)
    for (float& v : kernel) v /= sum;

    return kernel;
}

CUDAHandler* CUDAHandler::instance = nullptr;

CUDAHandler::CUDAHandler(int width, int height, GLuint textureID) :  width(width), height(height)
{
    hipGraphicsGLRegisterImage(&cudaResource, textureID, GL_TEXTURE_2D, hipGraphicsRegisterFlagsSurfaceLoadStore);
    instance = this; // store global reference (to be used for mouse and imGui User Interface (UI) operations)
}

CUDAHandler::~CUDAHandler()
{
    hipFree(d_leniaParticles); 
    hipFree(d_colors);
    hipFree(d_states);
    delete lenia;
      
    hipGraphicsUnregisterResource(cudaResource);
    
}
// _________________________________________________________________________//
void CUDAHandler::updateDraw(float dt)
{
    this->dt = dt;

    static Settings previousSettings;
    Settings currentSettings = {
        .numberOfParticles = totalParticles,
        .particleRadius = particleRadius,
        .spacing = spacing,
        .convRadius = convolutionRadius,
        .alpha = alpha,
        
    };
    if(leniaSize == 0 || currentSettings != previousSettings) {
        
        initLenia();
        previousSettings = currentSettings;
        
    }

    if(startSimulation){
        int kernelDiameter = 2 * convolutionRadius + 1;
        activate_LeniaGoL_convolution_kernel<<<gridSize, blockSize>>>(d_leniaParticles, leniaSize, lenia->gridRows, lenia->gridCols, kernelDiameter, convolutionRadius, sigma, mu, conv_dt);
        commitNextEnergy_kernel<<<gridSize, blockSize>>> (d_leniaParticles, leniaSize);
        thresholdAndCommit_kernel<<<gridSize, blockSize>>> (d_leniaParticles, leniaSize, d_colors, colorPallete.size());
    }
 
    hipSurfaceObject_t surface = MapSurfaceResouse();    
   
    clearGraphicsDisplay(surface, WHITE);
    // drawTriangle(surface, RED_MERCURY, vec2(100, 200), vec2(150, 600), vec2(500, 300));
    // checkCuda(hipPeekAtLastError());
    // checkCuda(hipDeviceSynchronize());
    // printf("again: %d\n", leniaSize);
    drawLeniaParticles<<<gridSize, blockSize>>>(surface, d_leniaParticles, leniaSize, width, height, 1.0f, 0.0f, 0.0f);

    // checkCuda(hipPeekAtLastError());
    // checkCuda(hipDeviceSynchronize());

    hipDestroySurfaceObject(surface);
    hipGraphicsUnmapResources(1, &cudaResource);
}

//________________________________________________________________________//

void CUDAHandler::clearGraphicsDisplay(hipSurfaceObject_t &surface, uchar4 color)
{
    int threads = 16; 
    dim3 clearBlock(threads, threads);
    dim3 clearGrid((width + clearBlock.x -1) / clearBlock.x, (height + clearBlock.y - 1) / clearBlock.y);
    clearSurface_kernel<<<clearGrid, clearBlock>>>(surface, width, height, color);
}

void CUDAHandler::drawTriangle(hipSurfaceObject_t &surface, uchar4 color, vec2 v0, vec2 v1, vec2 v2)
{

    // Sort vertices by y-coordinate
    if (v0.y > v1.y) { swap(v0, v1); }
    if (v0.y > v2.y) { swap(v0, v2); }
    if (v1.y > v2.y) { swap(v1, v2); }

    double yMax = v2.y;
    double yMin = v0.y;

    if (v0.x > v1.x) { swap(v0, v1);}
    if (v0.x > v2.x) { swap(v0, v2);}
    if (v1.x > v2.x) { swap(v1, v2);}

    double xMax = v2.x;
    double xMin = v0.x;

    // Calculate bounding box dimensions
    int boxWidth = ceilf(xMax) - floorf(xMin);
    int boxHeight = ceilf(yMax) - floorf(yMin);
    xMin = floorf(xMin);
    yMin = floorf(yMin);

    dim3 threads (16, 16);
    dim3 blocks((boxWidth + threads.x - 1)/ threads.x, (boxHeight + threads.y - 1)/ threads.y);
    drawTriangle_kernel<<<blocks, threads>>>(surface, width, height, RED_MERCURY, xMin, yMin, v0, v1, v2);


}

void CUDAHandler::initLenia()
{
    
    // Free previously allocated GPU memory if it exists
    if (d_leniaParticles) { 
        checkCuda(hipFree(d_leniaParticles));
        d_leniaParticles = nullptr;
    }
    if (d_states) {
        checkCuda(hipFree(d_states));
        d_states = nullptr;
    }
    if (d_colors) {
        checkCuda(hipFree(d_colors));
        d_colors = nullptr;
    }

    // Optional: delete previous Lenia object
    if (lenia) {
        delete lenia;
        lenia = nullptr;
    }

    std::vector<float> convKernel = generateCircularShellKernel(convolutionRadius, alpha);
    size_t bytes = convKernel.size() * sizeof(float);
    checkCuda(hipMemcpyToSymbol(HIP_SYMBOL(d_kernelConst), convKernel.data(), bytes));


    lenia = new Lenia(width, height, totalParticles, particleRadius, spacing, {16,10} );
    leniaSize = lenia->particles.size();
    


    blockSize = 256;
    gridSize = (leniaSize + blockSize - 1) / blockSize;
    
    checkCuda(hipMalloc(&d_leniaParticles, leniaSize * sizeof(Particle)));
    checkCuda(hipMemcpy(d_leniaParticles, lenia->particles.data(), leniaSize * sizeof(Particle), hipMemcpyHostToDevice));
    
    checkCuda(hipMalloc(&d_states, blockSize * gridSize * sizeof(hiprandState_t)));
    
    
    checkCuda(hipMalloc(&d_colors, colorPallete.size() * sizeof(uchar4)));
    checkCuda(hipMemcpy(d_colors, colorPallete.data(), colorPallete.size() * sizeof(uchar4), hipMemcpyHostToDevice));    
    
    init_random<<<gridSize, blockSize>>>(time(0), d_states);
    // checkCuda(hipPeekAtLastError());
    // checkCuda(hipDeviceSynchronize());
    // energy and colors
    initializeLeniaParticle<<<gridSize, blockSize>>>(d_leniaParticles, leniaSize, d_states, d_colors, colorPallete.size());
    // checkCuda(hipPeekAtLastError());
    // checkCuda(hipDeviceSynchronize());
    
    
}

hipSurfaceObject_t CUDAHandler::MapSurfaceResouse()
{
    //* Map the resource for CUDA
    hipArray_t array;
    
    hipGraphicsMapResources(1, &cudaResource, 0);
    hipGraphicsSubResourceGetMappedArray(&array, cudaResource, 0, 0);

    //* Create a CUDA surface object
    hipResourceDesc resDesc = {};
    resDesc.resType = hipResourceTypeArray;
    resDesc.res.array.array = array;

    hipSurfaceObject_t surface = 0;
    hipCreateSurfaceObject(&surface, &resDesc);
    return surface;
}
