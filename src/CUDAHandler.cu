#include "hip/hip_runtime.h"
#include "CUDAHandler.h"


__device__ float random_float_in_range(hiprandState_t* state, float a, float b) {
    // return a + (b - a) * curand_uniform_float(state);  // this does not include b  e.g -1 to 1.0  it does not include 1.0
    return a + (b - a) * (hiprand_uniform(state) - 0.5) * 2.0;  // this approach includes the upper limit   -1 to 1.0  it includes 1.0
}

__device__
void drawPixel(hipSurfaceObject_t surface, int x, int y, uchar4 color, int width, int height)
{
    if (x >= 0 && x < width && y >= 0 && y < height) {
        surf2Dwrite(color, surface, x * sizeof(uchar4), y);
    }
}

__device__ void drawFilledCircle(hipSurfaceObject_t surface, int cx, int cy, int radius, uchar4 color, int width, int height) {
    int rSquared = radius * radius;
    for (int dy = -radius; dy <= radius; ++dy) {
        int y = cy + dy;
        if (y < 0 || y >= height) continue;

        for (int dx = -radius; dx <= radius; ++dx) {
            int x = cx + dx;
            if (x < 0 || x >= width) continue;

            if (dx * dx + dy * dy <= rSquared) {
                drawPixel(surface, x, y, color, width, height);
            }
        }
    }
}


__global__ void init_random(unsigned int seed, hiprandState_t* states){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    hiprand_init(seed, idx, 0, &states[idx]);
}
__global__ void initializeLeniaParticle(Particle* particles, int totalParticles, hiprandState_t* states, uchar4* colors, int numberColors) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i >= totalParticles) return;
    hiprandState_t x = states[i];
    // float e =  random_float_in_range(&x, 0.1, 0.5);
    particles[i].energy = 1.5f;
    states[i] = x; // reinstate the random value;
    int colorIndex = static_cast<int>(1.0f * (numberColors - 1));
    colorIndex = max(0, min(colorIndex, numberColors - 1));
    particles[i].color = make_uchar4(0,0,0,255); // colors[colorIndex];
}

__global__ void drawLeniaParticles(hipSurfaceObject_t surface, Particle* particles, int numberParticles, int width, int height, float zoom, float panX, float panY){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= numberParticles) return;
    Particle p = particles[i];
    vec2 pos = p.position;
    if (threadIdx.x == 0){
        printf("position.x  %f\n", pos.x);
        printf("Energy %f\n", p.energy);
    }
    float radius = p.radius * zoom;
    int x0 = (int)(width / 2.0f + (pos.x + panX) * zoom);
    int y0 = (int)(height / 2.0f + (pos.y + panY) * zoom);
    
    drawFilledCircle(surface, x0, y0, radius, p.color, width, height);
}

__global__ void clearSurface_kernel(hipSurfaceObject_t surface, int width, int height, uchar4 color) {
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;

    if (x >= width || y >= height) return;
    surf2Dwrite(color, surface, x * sizeof(uchar4), y);
}



// __device__
// void drawPixel(hipSurfaceObject_t surface, int x, int y, uchar4 color, int width, int height)
// {
//     if (x >= 0 && x < width && y >= 0 && y < height) {
//         surf2Dwrite(color, surface, x * sizeof(uchar4), y);
//     }
// }

__device__ void drawLine(hipSurfaceObject_t surface, int x0, int y0, int x1, int y1, uchar4 color, int width, int height)
{
    int dx = abs(x1 - x0), sx = x0 < x1 ? 1 : -1;
    int dy = -abs(y1 - y0), sy = y0 < y1 ? 1 : -1;
    int err = dx + dy, e2;

    while (true) {
        drawPixel(surface, x0, y0, color, width, height);
        if (x0 == x1 && y0 == y1) break;
        e2 = 2 * err;
        if (e2 >= dy) { err += dy; x0 += sx; }
        if (e2 <= dx) { err += dx; y0 += sy; }
    }
}

__device__ void drawCircleOutline(hipSurfaceObject_t surface, int cx, int cy, int radius, uchar4 color, int width, int height) {
    const int segments = 36; // More segments = smoother circle
    for (int i = 0; i < segments; ++i) {
        float theta0 = (2.0f * M_PI * i) / segments;
        float theta1 = (2.0f * M_PI * (i + 1)) / segments;
        
        int x0 = cx + radius * cosf(theta0);
        int y0 = cy + radius * sinf(theta0);
        int x1 = cx + radius * cosf(theta1);
        int y1 = cy + radius * sinf(theta1);

        drawLine(surface, x0, y0, x1, y1, color, width, height);
    }
}

// __device__ void drawFilledCircle(hipSurfaceObject_t surface, int cx, int cy, int radius, uchar4 color, int width, int height) {
//     int rSquared = radius * radius;
//     for (int dy = -radius; dy <= radius; ++dy) {
//         int y = cy + dy;
//         if (y < 0 || y >= height) continue;

//         for (int dx = -radius; dx <= radius; ++dx) {
//             int x = cx + dx;
//             if (x < 0 || x >= width) continue;

//             if (dx * dx + dy * dy <= rSquared) {
//                 drawPixel(surface, x, y, color, width, height);
//             }
//         }
//     }
// }

__device__ bool threeEdgeTest2D(const vec2 &A, const vec2 &B, const vec2 &C, const vec2 &P)
{
    vec2 a = B - A;
    vec2 b = C - A;
    vec2 q = P - A;

    float cross_ab = a.x * b.y - a.y * b.x;

    if (cross_ab == 0.0f) return false; // Triangle is degenerate (no area)

    float s = (q.x * b.y - q.y * b.x) / cross_ab;
    float r = (a.x * q.y - a.y * q.x) / cross_ab;

    return s >= 0.0f && r >= 0.0f && (s + r) <= 1.0f;
}

__device__ __host__ void swap(vec2& a, vec2& b) {
        vec2 temp = a;
        a = b;
        b = temp;
}


__global__ void drawTriangle_kernel(hipSurfaceObject_t surface, int width, int height, uchar4 color, int minX, int minY, vec2 a, vec2 b, vec2 c)
{
    int local_x = threadIdx.x + blockIdx.x * blockDim.x;
    int local_y = threadIdx.y + blockIdx.y * blockDim.y;

    int x = local_x + minX;
    int y = local_y + minY;

    if (x >= width || y >= height) return;

    if (!threeEdgeTest2D(a, b, c, vec2(x, y))) return;
    drawPixel(surface, x, y, color, width, height);

    
}

CUDAHandler* CUDAHandler::instance = nullptr;

CUDAHandler::CUDAHandler(int width, int height, GLuint textureID) :  width(width), height(height)
{
    hipGraphicsGLRegisterImage(&cudaResource, textureID, GL_TEXTURE_2D, hipGraphicsRegisterFlagsSurfaceLoadStore);
    instance = this; // store global reference (to be used for mouse and imGui User Interface (UI) operations)
}

CUDAHandler::~CUDAHandler()
{
    hipFree(d_leniaParticles); 
    hipFree(d_colors);
    hipFree(d_states);
    delete lenia;
      
    hipGraphicsUnregisterResource(cudaResource);
    
}
// _________________________________________________________________________//
void CUDAHandler::updateDraw(float dt)
{
    this->dt = dt;

    if(leniaSize == 0) {
        initLenia();
        
    }
 
    hipSurfaceObject_t surface = MapSurfaceResouse();    
   
    clearGraphicsDisplay(surface, WHITE);
    // drawTriangle(surface, RED_MERCURY, vec2(100, 200), vec2(150, 600), vec2(500, 300));
    checkCuda(hipPeekAtLastError());
    checkCuda(hipDeviceSynchronize());
    // printf("again: %d\n", leniaSize);
    drawLeniaParticles<<<gridSize, blockSize>>>(surface, d_leniaParticles, leniaSize, width, height, 1.0f, 0.0f, 0.0f);

    checkCuda(hipPeekAtLastError());
    checkCuda(hipDeviceSynchronize());

    hipDestroySurfaceObject(surface);
    hipGraphicsUnmapResources(1, &cudaResource);
}

//________________________________________________________________________//

void CUDAHandler::clearGraphicsDisplay(hipSurfaceObject_t &surface, uchar4 color)
{
    int threads = 16; 
    dim3 clearBlock(threads, threads);
    dim3 clearGrid((width + clearBlock.x -1) / clearBlock.x, (height + clearBlock.y - 1) / clearBlock.y);
    clearSurface_kernel<<<clearGrid, clearBlock>>>(surface, width, height, color);
}

void CUDAHandler::drawTriangle(hipSurfaceObject_t &surface, uchar4 color, vec2 v0, vec2 v1, vec2 v2)
{

    // Sort vertices by y-coordinate
    if (v0.y > v1.y) { swap(v0, v1); }
    if (v0.y > v2.y) { swap(v0, v2); }
    if (v1.y > v2.y) { swap(v1, v2); }

    double yMax = v2.y;
    double yMin = v0.y;

    if (v0.x > v1.x) { swap(v0, v1);}
    if (v0.x > v2.x) { swap(v0, v2);}
    if (v1.x > v2.x) { swap(v1, v2);}

    double xMax = v2.x;
    double xMin = v0.x;

    // Calculate bounding box dimensions
    int boxWidth = ceilf(xMax) - floorf(xMin);
    int boxHeight = ceilf(yMax) - floorf(yMin);
    xMin = floorf(xMin);
    yMin = floorf(yMin);

    dim3 threads (16, 16);
    dim3 blocks((boxWidth + threads.x - 1)/ threads.x, (boxHeight + threads.y - 1)/ threads.y);
    drawTriangle_kernel<<<blocks, threads>>>(surface, width, height, RED_MERCURY, xMin, yMin, v0, v1, v2);


}

void CUDAHandler::initLenia()
{
    
    lenia = new Lenia(width, height, totalParticles, particleRadius, {16,10} );
    leniaSize = lenia->particles.size();
    // for(int i = 0; i < 10; i++){
    //     printf("position: (%f, %f) \n", lenia->particles[i]->position.x, lenia->particles[i]->position.y);
    // }
    blockSize = 256;
    gridSize = (leniaSize + blockSize - 1) / blockSize;
    
    checkCuda(hipMalloc(&d_leniaParticles, leniaSize * sizeof(Particle*)));
    checkCuda(hipMemcpy(d_leniaParticles, lenia->particles.data(), leniaSize * sizeof(Particle*), hipMemcpyHostToDevice));
    
    checkCuda(hipMalloc(&d_states, blockSize * gridSize * sizeof(hiprandState_t)));
    
    
    checkCuda(hipMalloc(&d_colors, colorPallete.size() * sizeof(uchar4)));
    checkCuda(hipMemcpy(d_colors, colorPallete.data(), colorPallete.size() * sizeof(uchar4), hipMemcpyHostToDevice));    
    
    init_random<<<gridSize, blockSize>>>(time(0), d_states);
    checkCuda(hipPeekAtLastError());
    checkCuda(hipDeviceSynchronize());
    // energy and colors
    initializeLeniaParticle<<<gridSize, blockSize>>>(d_leniaParticles, leniaSize, d_states, d_colors, colorPallete.size());
    checkCuda(hipPeekAtLastError());
    checkCuda(hipDeviceSynchronize());
    printf("LeniaSize: %d\n", leniaSize);
    
}

hipSurfaceObject_t CUDAHandler::MapSurfaceResouse()
{
    //* Map the resource for CUDA
    hipArray_t array;
    // glFinish();
    hipGraphicsMapResources(1, &cudaResource, 0);
    hipGraphicsSubResourceGetMappedArray(&array, cudaResource, 0, 0);

    //* Create a CUDA surface object
    hipResourceDesc resDesc = {};
    resDesc.resType = hipResourceTypeArray;
    resDesc.res.array.array = array;

    hipSurfaceObject_t surface = 0;
    hipCreateSurfaceObject(&surface, &resDesc);
    return surface;
}
