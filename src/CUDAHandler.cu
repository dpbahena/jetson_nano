#include "hip/hip_runtime.h"
#include "CUDAHandler.h"
#include <algorithm>


constexpr int MAX_RADIUS = 12;                    // change to suit your max
constexpr int MAX_K      = (2*MAX_RADIUS+1)*(2*MAX_RADIUS+1);

__constant__ float d_kernelConst[MAX_K];          // <── NOT a pointer



__device__ float random_float_in_range(hiprandState_t* state, float a, float b) {
    // return a + (b - a) * curand_uniform_float(state);  // this does not include b  e.g -1 to 1.0  it does not include 1.0
    return a + (b - a) * (hiprand_uniform(state) - 0.5) * 2.0;  // this approach includes the upper limit   -1 to 1.0  it includes 1.0
}

__device__
void drawPixel(hipSurfaceObject_t surface, int x, int y, uchar4 color, int width, int height)
{
    if (x >= 0 && x < width && y >= 0 && y < height) {
        surf2Dwrite(color, surface, x * sizeof(uchar4), y);
    }
}

__device__ void drawFilledCircle(hipSurfaceObject_t surface, int cx, int cy, int radius, uchar4 color, int width, int height) {
    int rSquared = radius * radius;
    for (int dy = -radius; dy <= radius; ++dy) {
        int y = cy + dy;
        if (y < 0 || y >= height) continue;

        for (int dx = -radius; dx <= radius; ++dx) {
            int x = cx + dx;
            if (x < 0 || x >= width) continue;

            if (dx * dx + dy * dy <= rSquared) {
                drawPixel(surface, x, y, color, width, height);
            }
        }
    }
}


__global__ void init_random(unsigned int seed, hiprandState_t* states){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    hiprand_init(seed, idx, 0, &states[idx]);
}
__global__ void initializeLeniaParticle(Particle* particles, int totalParticles, hiprandState_t* states, uchar4* colors, int numberColors) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i >= totalParticles) return;
    hiprandState_t x = states[i];
    // float e =  random_float_in_range(&x, 0.1, 0.5);
    // initial noise
    float e = hiprand_uniform(&x) * 0.5f;
    // float e = hiprand_uniform(&x) < 0.15f ? hiprand_uniform(&x) * 0.35f : 0.0f; 
    particles[i].energy = e;
    states[i] = x; // reinstate the random value;
    int colorIndex = static_cast<int>(e * (numberColors - 1));
    colorIndex = max(0, min(colorIndex, numberColors - 1));
    particles[i].color = colors[colorIndex];
}

// __global__ void drawLeniaParticles(hipSurfaceObject_t surface, Particle* particles, int numberParticles, int width, int height, float zoom, float panX, float panY){
//     int i = blockIdx.x * blockDim.x + threadIdx.x;
//     if (i >= numberParticles) return;
//     Particle p = particles[i];
//     vec2 pos = p.position;
//     // if (threadIdx.x == 0){
//     //     printf("position.x  %f\n", pos.x);
//     //     printf("Energy %f\n", p.energy);
//     // }
//     float radius = p.radius * zoom;
//     int x0 = (int)(width / 2.0f + (pos.x + panX) * zoom);
//     int y0 = (int)(height / 2.0f + (pos.y + panY) * zoom);
    
//     drawFilledCircle(surface, x0, y0, radius, p.color, width, height);
// }

__global__ void commitNextEnergy_kernel(Particle* particles, int totalParticles) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i >= totalParticles) return;

    particles[i].energy = particles[i].nextEnergy;
    particles[i].nextEnergy = 0;
}

__global__ void thresholdAndCommit_kernel(Particle* g,
                                          int n,
                                          const uchar4* colors,
                                          int numColors,
                                          float thresh=0.2f /* e.g. 0.3f */)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i >= n) return;

    const float e = g[i].energy;          // already up-to-date from step-2

    // 1.  Boolean life flag (only used if you still need a discrete state)
    g[i].alive = (e > thresh);

    // 2.  Colour lookup for rendering – linear ramp through your palette
    int idx = (int)(e * (numColors - 1) + 0.5f);
    idx = max(0, min(idx, numColors - 1));
    g[i].color = colors[idx];
}

// __device__ float growthMapping(float u, float mu, float sigma) {
//     return 2.0f * expf(-powf((u - mu), 2) / (2.0f * sigma * sigma)) - 1.0f;
// }
__device__ float myGrowthMapping(float u, float mu, float sigma) {
    float z = ((u - mu) * (u - mu)) / ( 2 * sigma * sigma);
    return 2 * expf(-z) - 1.0f;
}
__device__ float growthMapping(float u, float mu, float sigma) {
    float z = (u - mu) / sigma;
    return expf(-0.5f * z * z) * 2.0f - 1.0f;
}
__device__ float growthMappingquad4(float u, float mu, float sigma) {
    float x = 1.0f - ((u - mu) * (u - mu)) / (9.0f * sigma * sigma);
    x = max(0.0f, x);
    return std::pow(x, 4) * 2.0f - 1.0f;
}

__device__ float growth_quad4(float u, float mu, float sigma) {
    float x = 1.0f - ((u - mu) * (u - mu)) / (9.0f * sigma * sigma);
    x = fmaxf(0.0f, x);
    return powf(x, 4) * 2.0f - 1.0f;
}
__device__ float growth_gaussian(float u, float mu, float sigma) {
    float x = (u - mu) / sigma;
    return 2.0f * expf(-x * x * 4.5f) - 1.0f;
}


__device__ float growth_triangle(float u, float mu, float sigma) {
    float dist = fabsf(u - mu);
    float x = 1.0f - dist / (1.5f * sigma);
    return fmaxf(0.0f, x * 2.0f - 1.0f);
}
__device__ float growth_step(float u, float mu, float sigma) {
    return (fabsf(u - mu) < sigma) ? 1.0f : -1.0f;
}
__device__ float growth_relu(float u, float mu, float sigma) {
    float x = 1.0f - fabsf(u - mu) / sigma;
    return fmaxf(0.0f, x) * 2.0f - 1.0f;
}




// __global__ void activate_LeniaGoL_convolution_kernel(
//     Particle* particles,
//     int totalParticles,
//     int gridRows,
//     int gridCols,
//     int kernelDiameter,
//     int radius,
//     float sigma,
//     float mu,
//     float dt
// ) {
//     int i = threadIdx.x + blockIdx.x * blockDim.x;
//     if (i >= totalParticles) return;

//     int row = i / gridCols;
//     int col = i % gridCols;

//     float neighborSum = 0.0f;
//     float weightSum = 0.0f;

//     for (int dr = -radius; dr <= radius; ++dr) {
//         for (int dc = -radius; dc <= radius; ++dc) {
//             int nr = row + dr;
//             int nc = col + dc;
//             // float distance = sqrtf(dr * dr + dc * dc);
//             if (nr >= 0 && nr < gridRows && nc >= 0 && nc < gridCols) {
//                 int j = nr * gridCols + nc;
//                 // float weight = gaussianKernel(distance / radius, sigma);
//                 int kernelIndex = (dr + radius) * kernelDiameter + (dc + radius);
//                 float weight = d_kernelConst[kernelIndex];
//                 neighborSum += particles[j].energy * weight;
//                 weightSum += weight;
//             }
//         }
//     }

   
//     float u = neighborSum  / weightSum ;
    

//     float growth = growthMapping(u, mu, sigma);
//     float e = fminf(1.0f, fmaxf(0.0f, particles[i].energy + dt * growth));
//     particles[i].nextEnergy = e;
// }



__global__ void drawLeniaParticles(hipSurfaceObject_t surface, Particle* particles, int numberParticles, int width, int height, float zoom, float panX, float panY){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= numberParticles ) return;

    Particle p = particles[i];

    // Apply zoom and pan directly (without adding screen center shift)
    float radius = p.radius * zoom;
    int x0 = (int)((p.position.x + panX) * zoom);
    int y0 = (int)((p.position.y + panY) * zoom);

    // Reject particles that fall outside the surface
    if (x0 < 0 || x0 >= width || y0 < 0 || y0 >= height) return;

    drawFilledCircle(surface, x0, y0, radius, p.color, width, height);
}

__global__ void activate_LeniaGoL_convolution_kernel(
    Particle* particles,
    int totalParticles,
    int gridRows,
    int gridCols,
    int kernelDiameter,
    int radius,
    float sigma,
    float mu,
    float dt 
) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i >= totalParticles) return;

    int row = i / gridCols;
    int col = i % gridCols;

    float neighborSum = 0.0f;
    float weightSum = 0.0f;

    for (int dr = -radius; dr <= radius; ++dr) {
        for (int dc = -radius; dc <= radius; ++dc) {
            // Wrap around (periodic boundary)
            int nr = (row + dr + gridRows) % gridRows;
            int nc = (col + dc + gridCols) % gridCols;

            int j = nr * gridCols + nc;
            int kernelIndex = (dr + radius) * kernelDiameter + (dc + radius);

            float weight = d_kernelConst[kernelIndex];
            neighborSum += particles[j].energy * weight;
            weightSum += weight;
        }
    }

    // float u = (weightSum > 0.0f) ? (neighborSum / weightSum) : 0.0f;
    float u = neighborSum;
    float growth = growthMapping(u, mu, sigma);
    float e = fminf(1.0f, fmaxf(0.0f, particles[i].energy + dt * growth));
    // TEMP: Visualize normalized excitation and growth directly
    uchar4 debugColor;
    debugColor.x = (unsigned char)(255.0f * fminf(fmaxf(u, 0.0f), 1.0f));        // Red = excitation u
    debugColor.y = (unsigned char)(255.0f * fminf(fmaxf((growth + 1.0f) * 0.5f, 0.0f), 1.0f)); // Green = growth (-1 to 1 mapped to 0-1)
    debugColor.z = 0;
    debugColor.w = 255;

    particles[i].color = debugColor;

    particles[i].nextEnergy = e;
   

}

__global__ void activate_LeniaGoL_convolution_kernel(
    Particle* particles,
    int totalParticles,
    int gridRows,
    int gridCols,
    int kernelDiameter,
    int radius,
    float sigma,
    float mu,
    float dt,
    float* debugU,
    float* debugGrowth 
) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i >= totalParticles) return;

    int row = i / gridCols;
    int col = i % gridCols;

    float neighborSum = 0.0f;
    float weightSum = 0.0f;

    for (int dr = -radius; dr <= radius; ++dr) {
        for (int dc = -radius; dc <= radius; ++dc) {
            // Wrap around (periodic boundary)
            int nr = (row + dr + gridRows) % gridRows;
            int nc = (col + dc + gridCols) % gridCols;

            int j = nr * gridCols + nc;
            int kernelIndex = (dr + radius) * kernelDiameter + (dc + radius);

            float weight = d_kernelConst[kernelIndex];
            neighborSum += particles[j].energy * weight;
            weightSum += weight;
        }
    }

    float u = (weightSum > 0.0f) ? (neighborSum / weightSum) : 0.0f;

    float growth = myGrowthMapping(u, mu, sigma);
    
    // float growth = growthMappingquad4(u, mu, sigma);
    // float growth = growth_triangle(u, mu, sigma);
    // float growth = growth_gaussian(u, mu, sigma);
    // float growth = growth_relu(u, mu, sigma);
    // float growth = growth_step(u, mu, sigma);
    float e = fminf(1.0f, fmaxf(0.0f, particles[i].energy + dt * growth));
    // TEMP: Visualize normalized excitation and growth directly
    // uchar4 debugColor;
    // debugColor.x = (unsigned char)(255.0f * fminf(fmaxf(u, 0.0f), 1.0f));        // Red = excitation u
    // debugColor.y = (unsigned char)(255.0f * fminf(fmaxf((growth + 1.0f) * 0.5f, 0.0f), 1.0f)); // Green = growth (-1 to 1 mapped to 0-1)
    // debugColor.z = 0;
    // debugColor.w = 255;

    // particles[i].color = debugColor;

    particles[i].nextEnergy = e;
    // if (i == 0) {
    //     // Store a sample (first thread only — for simplicity)
    //     debugU[0] = u;
    //     debugGrowth[0] = growth;
    // }
    if (i == 0) {
        if (!isnan(u) && !isnan(growth)) {
            debugU[0] = u;
            debugGrowth[0] = growth;
        } else {
            debugU[0] = 0.0f;
            debugGrowth[0] = 0.0f;
        }
    }
    // if (i == 0) {
    //     printf("Sample u = %.5f, growth = %.5f\n", u, growth);
    // }


   

}


__global__ void clearSurface_kernel(hipSurfaceObject_t surface, int width, int height, uchar4 color) {
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;

    if (x >= width || y >= height) return;
    surf2Dwrite(color, surface, x * sizeof(uchar4), y);
}



// __device__
// void drawPixel(hipSurfaceObject_t surface, int x, int y, uchar4 color, int width, int height)
// {
//     if (x >= 0 && x < width && y >= 0 && y < height) {
//         surf2Dwrite(color, surface, x * sizeof(uchar4), y);
//     }
// }

__device__ void drawLine(hipSurfaceObject_t surface, int x0, int y0, int x1, int y1, uchar4 color, int width, int height)
{
    int dx = abs(x1 - x0), sx = x0 < x1 ? 1 : -1;
    int dy = -abs(y1 - y0), sy = y0 < y1 ? 1 : -1;
    int err = dx + dy, e2;

    while (true) {
        drawPixel(surface, x0, y0, color, width, height);
        if (x0 == x1 && y0 == y1) break;
        e2 = 2 * err;
        if (e2 >= dy) { err += dy; x0 += sx; }
        if (e2 <= dx) { err += dx; y0 += sy; }
    }
}

__device__ void drawCircleOutline(hipSurfaceObject_t surface, int cx, int cy, int radius, uchar4 color, int width, int height) {
    const int segments = 36; // More segments = smoother circle
    for (int i = 0; i < segments; ++i) {
        float theta0 = (2.0f * M_PI * i) / segments;
        float theta1 = (2.0f * M_PI * (i + 1)) / segments;
        
        int x0 = cx + radius * cosf(theta0);
        int y0 = cy + radius * sinf(theta0);
        int x1 = cx + radius * cosf(theta1);
        int y1 = cy + radius * sinf(theta1);

        drawLine(surface, x0, y0, x1, y1, color, width, height);
    }
}

// __device__ void drawFilledCircle(hipSurfaceObject_t surface, int cx, int cy, int radius, uchar4 color, int width, int height) {
//     int rSquared = radius * radius;
//     for (int dy = -radius; dy <= radius; ++dy) {
//         int y = cy + dy;
//         if (y < 0 || y >= height) continue;

//         for (int dx = -radius; dx <= radius; ++dx) {
//             int x = cx + dx;
//             if (x < 0 || x >= width) continue;

//             if (dx * dx + dy * dy <= rSquared) {
//                 drawPixel(surface, x, y, color, width, height);
//             }
//         }
//     }
// }

__device__ bool threeEdgeTest2D(const vec2 &A, const vec2 &B, const vec2 &C, const vec2 &P)
{
    vec2 a = B - A;
    vec2 b = C - A;
    vec2 q = P - A;

    float cross_ab = a.x * b.y - a.y * b.x;

    if (cross_ab == 0.0f) return false; // Triangle is degenerate (no area)

    float s = (q.x * b.y - q.y * b.x) / cross_ab;
    float r = (a.x * q.y - a.y * q.x) / cross_ab;

    return s >= 0.0f && r >= 0.0f && (s + r) <= 1.0f;
}

__device__ __host__ void swap(vec2& a, vec2& b) {
        vec2 temp = a;
        a = b;
        b = temp;
}


__global__ void drawTriangle_kernel(hipSurfaceObject_t surface, int width, int height, uchar4 color, int minX, int minY, vec2 a, vec2 b, vec2 c)
{
    int local_x = threadIdx.x + blockIdx.x * blockDim.x;
    int local_y = threadIdx.y + blockIdx.y * blockDim.y;

    int x = local_x + minX;
    int y = local_y + minY;

    if (x >= width || y >= height) return;

    if (!threeEdgeTest2D(a, b, c, vec2(x, y))) return;
    drawPixel(surface, x, y, color, width, height);

    
}


// ────────────────────────────────
//  Chan-Lenia “shell” kernel
//  (build once on the host)
// ────────────────────────────────
std::vector<float> CUDAHandler::generateCircularShellKernel(int radius, float alpha /* ≈4 */)
{
    int  diameter = 2 * radius + 1;
    std::vector<float> kernel(diameter * diameter, 0.0f);

    float sum = 0.0f;
    for (int y = -radius; y <= radius; ++y)
        for (int x = -radius; x <= radius; ++x)
        {
            float dist = std::sqrt(float(x*x + y*y));     // Euclidean distance
            if (dist > radius) continue;                  // outside the circle

            float r = dist / radius;                      // normalise to [0,1]
            float value = std::exp(alpha - alpha / (4.0f * r * (1.0f - r)));
            // ────────────────▲  shell(r, alpha)
            // float value = std::exp(-(r * r) / (2.0f * alpha * alpha));
            // float value = std::exp(alpha - (alpha/(4 * r * (1 - r))));
            // float value = std::pow(4 * r * (1 - r), alpha);
            // float value = std::exp(alpha -1.0f / (2.0f * r * r));

            kernel[(y + radius) * diameter + (x + radius)] = value;
            sum += value;
        }

    // normalise so ΣK = 1 (preserves total mass)
    for (float& v : kernel) v /= sum;

    return kernel;
}

std::vector<float> CUDAHandler::generateCircularBellKernel(int radius, float m, float s)
{
    kernelSize = 2 * radius + 1;
    std::vector<float> kernel(kernelSize * kernelSize, 0.0f);

    float sum = 0.0f;
    for (int y = -radius; y <= radius; ++y)
    {
        for (int x = -radius; x <= radius; ++x)
        {
            float dist = std::sqrt(float(x * x + y * y)); // Euclidean distance
            if (dist > radius) continue;                 // outside the circle

            float r = dist / radius;                     // normalize to [0, 1]

            // --- Lenia-style bell curve ---
            // float value = 2.0f *  std::exp(-((r - m) * (r - m)) / (2.0f * s * s)) - 1.f;
            float value = std::exp(-((r - m) * (r - m)) / (2.0f * s * s));

            kernel[(y + radius) * kernelSize + (x + radius)] = value;
            sum += value;
        }
    }

    // normalize so that sum(kernel) = 1
    for (float& v : kernel) v /= sum;

    return kernel;
}


CUDAHandler* CUDAHandler::instance = nullptr;

CUDAHandler::CUDAHandler(int width, int height, GLuint textureID) :  width(width), height(height)
{
    hipGraphicsGLRegisterImage(&cudaResource, textureID, GL_TEXTURE_2D, hipGraphicsRegisterFlagsSurfaceLoadStore);
    instance = this; // store global reference (to be used for mouse and imGui User Interface (UI) operations)
}

CUDAHandler::~CUDAHandler()
{
    hipFree(d_leniaParticles); 
    hipFree(d_colors);
    hipFree(d_states);
    hipFree(d_debugGrowth);
    hipFree(d_debugU);
    delete lenia;
      
    hipGraphicsUnregisterResource(cudaResource);
    
}

// std::vector<float> CUDAHandler::uHistory;
// std::vector<float> CUDAHandler::growthHistory;
// _________________________________________________________________________//
void CUDAHandler::updateDraw(float dt)
{
    this->dt = dt;

    static Settings previousSettings;
    Settings currentSettings = {
        .numberOfParticles = totalParticles,
        .particleRadius = particleRadius,
        .spacing = spacing,
        .convRadius = convolutionRadius,
        .alpha = alpha,
        .sigma = sigma,
        .mu = mu,
        .m = m,
        .s = s,
        .conv_dt = conv_dt
        
    };
    if(leniaSize == 0 || currentSettings != previousSettings) {
        
        initLenia();
        previousSettings = currentSettings;
        
    }
    

    if(startSimulation){
        int kernelDiameter = 2 * convolutionRadius + 1;
        activate_LeniaGoL_convolution_kernel<<<gridSize, blockSize>>>(d_leniaParticles, leniaSize, lenia->gridRows, lenia->gridCols, kernelDiameter, convolutionRadius, sigma, mu, conv_dt, d_debugU, d_debugGrowth);
        commitNextEnergy_kernel<<<gridSize, blockSize>>> (d_leniaParticles, leniaSize);
        thresholdAndCommit_kernel<<<gridSize, blockSize>>> (d_leniaParticles, leniaSize, d_colors, colorPallete.size());
        checkCuda(hipDeviceSynchronize());
        checkCuda(hipMemcpy(&debugU_host, d_debugU, sizeof(float), hipMemcpyDeviceToHost));
        checkCuda(hipMemcpy(&debugGrowth_host, d_debugGrowth, sizeof(float), hipMemcpyDeviceToHost));
        // Store it in a rolling buffer
        if (uHistory.size() > 100) uHistory.erase(uHistory.begin());
        uHistory.push_back(debugU_host);
     
        if (growthHistory.size() > 100) growthHistory.erase(growthHistory.begin());
        growthHistory.push_back(debugGrowth_host);


           
    }
 
    hipSurfaceObject_t surface = MapSurfaceResourse();   

    

    clearGraphicsDisplay(surface, WHITE);
    // drawTriangle(surface, RED_MERCURY, vec2(100, 200), vec2(150, 600), vec2(500, 300));
    // checkCuda(hipPeekAtLastError());
    // checkCuda(hipDeviceSynchronize());
    // printf("again: %d\n", leniaSize);
    drawLeniaParticles<<<gridSize, blockSize>>>(surface, d_leniaParticles, leniaSize, width, height, 1.0f, 0.0f, 0.0f);

    checkCuda(hipPeekAtLastError());
    checkCuda(hipDeviceSynchronize());

   
    hipDestroySurfaceObject(surface);
    hipGraphicsUnmapResources(1, &cudaResource);
}

//________________________________________________________________________//

void CUDAHandler::clearGraphicsDisplay(hipSurfaceObject_t &surface, uchar4 color)
{
    int threads = 16; 
    dim3 clearBlock(threads, threads);
    dim3 clearGrid((width + clearBlock.x -1) / clearBlock.x, (height + clearBlock.y - 1) / clearBlock.y);
    clearSurface_kernel<<<clearGrid, clearBlock>>>(surface, width, height, color);
}

void CUDAHandler::drawTriangle(hipSurfaceObject_t &surface, uchar4 color, vec2 v0, vec2 v1, vec2 v2)
{

    // Sort vertices by y-coordinate
    if (v0.y > v1.y) { swap(v0, v1); }
    if (v0.y > v2.y) { swap(v0, v2); }
    if (v1.y > v2.y) { swap(v1, v2); }

    double yMax = v2.y;
    double yMin = v0.y;

    if (v0.x > v1.x) { swap(v0, v1);}
    if (v0.x > v2.x) { swap(v0, v2);}
    if (v1.x > v2.x) { swap(v1, v2);}

    double xMax = v2.x;
    double xMin = v0.x;

    // Calculate bounding box dimensions
    int boxWidth = ceilf(xMax) - floorf(xMin);
    int boxHeight = ceilf(yMax) - floorf(yMin);
    xMin = floorf(xMin);
    yMin = floorf(yMin);

    dim3 threads (16, 16);
    dim3 blocks((boxWidth + threads.x - 1)/ threads.x, (boxHeight + threads.y - 1)/ threads.y);
    drawTriangle_kernel<<<blocks, threads>>>(surface, width, height, RED_MERCURY, xMin, yMin, v0, v1, v2);


}

void CUDAHandler::drawCircularKernel(hipSurfaceObject_t &surface)
{

    // Calculate bounding box   // if not zoom , nor panx, nor pany involved
    // vec2 position(300, 300);
    
    // int xMin = max(0, (int)(position.x - convolutionRadius));
    // int xMax = min(width - 1, (int)(position.x + convolutionRadius));
    // int yMin = max(0, (int)(position.y - convolutionRadius));
    // int yMax = min(height - 1, (int)(position.y + convolutionRadius));

    // int drawWidth   = xMax - xMin + 1;
    // int drawHeight  = yMax - yMin + 1;

    // dim3 blockSize(16, 16);
    // dim3 gridSize ((drawWidth + blockSize.x - 1) / blockSize.x, (drawHeight + blockSize.y -1) / blockSize.y);
    // drawGlowingCircle_kernel<<<gridSize, blockSize>>>(surface, width, height, position.x, position.y, convolutionRadius,  color, 1.5f, xmin, ymin, zoom, panX, panY);
}

void CUDAHandler::initLenia()
{
    
    // Free previously allocated GPU memory if it exists
    if (d_leniaParticles) { 
        checkCuda(hipFree(d_leniaParticles));
        d_leniaParticles = nullptr;
    }
    if (d_states) {
        checkCuda(hipFree(d_states));
        d_states = nullptr;
    }
    if (d_colors) {
        checkCuda(hipFree(d_colors));
        d_colors = nullptr;
    }
    if (d_debugGrowth) {
        checkCuda(hipFree(d_debugGrowth));
        d_debugGrowth = nullptr;
    }
    if (d_debugU) {
        checkCuda(hipFree(d_debugU));
        d_debugU = nullptr;
    }


    // Optional: delete previous Lenia object
    if (lenia) {
        delete lenia;
        lenia = nullptr;
    }

    // convKernel = generateCircularShellKernel(convolutionRadius, alpha);
    convKernel = generateCircularBellKernel(convolutionRadius, mu, sigma);

     
    
    size_t bytes = convKernel.size() * sizeof(float);
    checkCuda(hipMemcpyToSymbol(HIP_SYMBOL(d_kernelConst), convKernel.data(), bytes));
    int kernelSize = 2 * convolutionRadius + 1;
    imageData.resize(kernelSize * kernelSize * 4);

    float minVal = *std::min_element(convKernel.begin(), convKernel.end());
    float maxVal = *std::max_element(convKernel.begin(), convKernel.end());


    for (int y = 0; y < kernelSize; ++y) {
        for (int x = 0; x < kernelSize; ++x) {
            float value = convKernel[y * kernelSize + x];  // Assume normalized in [0, 1]
            // value = fminf(fmaxf(value, 0.0f), 1.0f);          // Clamp

            // // Map to a color (e.g., blue–yellow gradient)
            // float r = value;
            // float g = value * 0.6f + 0.4f; // emphasize mid values
            // float b = 1.0f - value;

            // int idx = (y * kernelSize + x) * 4;
            // imageData[idx + 0] = static_cast<unsigned char>(r * 255);
            // imageData[idx + 1] = static_cast<unsigned char>(g * 255);
            // imageData[idx + 2] = static_cast<unsigned char>(b * 255);
            // imageData[idx + 3] = 255;  // Alpha

            
            value = (value - minVal) / (maxVal - minVal + 1e-8f);  // Normalize locally

            vec3 rgb = turboColorMap(value);
            int idx = (y * kernelSize + x) * 4;

            imageData[idx + 0] = static_cast<unsigned char>(rgb.x * 255);
            imageData[idx + 1] = static_cast<unsigned char>(rgb.y * 255);
            imageData[idx + 2] = static_cast<unsigned char>(rgb.z * 255);
            imageData[idx + 3] = 255;

        }
    }

    lenia = new Lenia(width, height, totalParticles, particleRadius, spacing, {16,10} );
    leniaSize = lenia->particles.size();
    


    blockSize = 256;
    gridSize = (leniaSize + blockSize - 1) / blockSize;
    
    checkCuda(hipMalloc(&d_leniaParticles, leniaSize * sizeof(Particle)));
    checkCuda(hipMemcpy(d_leniaParticles, lenia->particles.data(), leniaSize * sizeof(Particle), hipMemcpyHostToDevice));
    
    checkCuda(hipMalloc(&d_states, blockSize * gridSize * sizeof(hiprandState_t)));
    
    
    checkCuda(hipMalloc(&d_colors, colorPallete.size() * sizeof(uchar4)));
    checkCuda(hipMemcpy(d_colors, colorPallete.data(), colorPallete.size() * sizeof(uchar4), hipMemcpyHostToDevice));    
    


    init_random<<<gridSize, blockSize>>>(time(0), d_states);
    // checkCuda(hipPeekAtLastError());
    // checkCuda(hipDeviceSynchronize());
    // energy and colors
    initializeLeniaParticle<<<gridSize, blockSize>>>(d_leniaParticles, leniaSize, d_states, d_colors, colorPallete.size());
    // checkCuda(hipPeekAtLastError());
    // checkCuda(hipDeviceSynchronize());

    checkCuda(hipMalloc(&d_debugU, sizeof(float)));
    checkCuda(hipMalloc(&d_debugGrowth, sizeof(float)));
    
    
}

hipSurfaceObject_t CUDAHandler::MapSurfaceResourse()
{
    //* Map the resource for CUDA
    hipArray_t array;
    
    hipGraphicsMapResources(1, &cudaResource, 0);
    hipGraphicsSubResourceGetMappedArray(&array, cudaResource, 0, 0);

    //* Create a CUDA surface object
    hipResourceDesc resDesc = {};
    resDesc.resType = hipResourceTypeArray;
    resDesc.res.array.array = array;

    hipSurfaceObject_t surface = 0;
    hipCreateSurfaceObject(&surface, &resDesc);
    return surface;
}

vec3 CUDAHandler::turboColorMap(float x)
{
    x = fminf(fmaxf(x, 0.0f), 1.0f);  // Clamp to [0,1]

    const float r = 34.61f + x * (1172.33f + x * (-10793.56f + x * (33300.12f + x * (-38394.49f + x * 14825.05f))));
    const float g = 23.31f + x * (557.33f + x * (1572.88f + x * (-7743.36f + x * (10332.56f + x * -3584.14f))));
    const float b = 27.2f  + x * (3211.1f + x * (-15327.97f + x * (27814.0f + x * (-22569.18f + x * 6838.66f))));

    return vec3(r, g, b) / 255.0f;
}